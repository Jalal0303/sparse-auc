#include "hip/hip_runtime.h"
//
// Created by baojian on 12/19/19.
//
# include <hip/hip_runtime.h>
# include "hipblas.h"

#define sign(x) (x > 0) - (x < 0)

typedef struct {
    double val;
    int index;
} data_pair;

static inline int __comp_descend(const void *a, const void *b) {
    if (((data_pair *) a)->val < ((data_pair *) b)->val) {
        return 1;
    } else {
        return -1;
    }
}

void _arg_sort_descend(const double *x, int *sorted_indices, int x_len) {
    data_pair *w_pairs = malloc(sizeof(data_pair) * x_len);
    for (int i = 0; i < x_len; i++) {
        w_pairs[i].val = x[i];
        w_pairs[i].index = i;
    }
    qsort(w_pairs, (size_t) x_len, sizeof(data_pair), &__comp_descend);
    for (int i = 0; i < x_len; i++) {
        sorted_indices[i] = w_pairs[i].index;
    }
    free(w_pairs);
}

double _auc_score(const double *true_labels, const double *scores, int len) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double *fpr = malloc(sizeof(double) * (len + 1));
    double *tpr = malloc(sizeof(double) * (len + 1));
    double num_posi = 0.0;
    double num_nega = 0.0;
    for (int i = 0; i < len; i++) {
        if (true_labels[i] > 0) {
            num_posi++;
        } else {
            num_nega++;
        }
    }
    int *sorted_indices = malloc(sizeof(int) * len);
    _arg_sort_descend(scores, sorted_indices, len);
    tpr[0] = 0.0; // initial point.
    fpr[0] = 0.0; // initial point.
    // accumulate sum
    for (int i = 0; i < len; i++) {
        double cur_label = true_labels[sorted_indices[i]];
        if (cur_label > 0) {
            fpr[i + 1] = fpr[i];
            tpr[i + 1] = tpr[i] + 1.0;
        } else {
            fpr[i + 1] = fpr[i] + 1.0;
            tpr[i + 1] = tpr[i];
        }
    }
    // cblas_dscal(len, 1. / num_posi, tpr, 1);
    // cblas_dscal(len, 1. / num_nega, fpr, 1);
    double alpha = 1. / num_posi;
    hipMallocManaged(&tpr, len * sizeof(double));
    hipblasDscal(handle, len, &alpha, tpr, 1);
    alpha = 1. / num_nega;
    hipMallocManaged(&fpr, len * sizeof(double));
    hipblasDscal(handle, len, &alpha, fpr, 1);
    hipDeviceSynchronize();
    //AUC score
    double auc = 0.0;
    double prev = 0.0;
    for (int i = 0; i < len; i++) {
        auc += (tpr[i] * (fpr[i] - prev));
        prev = fpr[i];
    }
    free(sorted_indices);
    free(fpr);
    free(tpr);
    hipblasDestroy(handle);
    return auc;
}

void _algo_spam_sparse(
        const double *x_tr_vals, const int *x_tr_inds, const int *x_tr_poss, const int *x_tr_lens,
        const double *data_y_tr, int data_n, int data_p, double para_c, double para_l1_reg,
        double para_l2_reg, int para_num_passes, int para_step_len, int para_reg_opt,
        int para_verbose, double *re_wt, double *re_wt_bar, double *re_auc, double *re_rts,
        int *re_len_auc) {

    double start_time = clock();
    double *grad_wt = malloc(sizeof(double) * data_p); // gradient
    double a_wt, *posi_x_mean = calloc((size_t) data_p, sizeof(double)); // w^T*E[x|y=1]
    double b_wt, *nega_x_mean = calloc((size_t) data_p, sizeof(double)); // w^T*E[x|y=-1]
    double alpha_wt, posi_t = 0.0, nega_t = 0.0;
    double *y_pred = calloc((size_t) data_n, sizeof(double));
    for (int i = 0; i < data_n; i++) {
        const int *xt_inds = x_tr_inds + x_tr_poss[i];
        const double *xt_vals = x_tr_vals + x_tr_poss[i];
        if (data_y_tr[i] > 0) {
            posi_t++;
            for (int kk = 0; kk < x_tr_lens[i]; kk++)
                posi_x_mean[xt_inds[kk]] += xt_vals[kk];
        } else {
            nega_t++;
            for (int kk = 0; kk < x_tr_lens[i]; kk++)
                nega_x_mean[xt_inds[kk]] += xt_vals[kk];
        }
    }
    double tmp1 = 1. / posi_t, tmp2 = 1. / nega_t;
    //cblas_dscal(data_p, 1. / posi_t, posi_x_mean, 1);
    //cblas_dscal(data_p, 1. / nega_t, nega_x_mean, 1);
    hipblasHandle_t handle;
    hipMallocManaged(&posi_x_mean, data_p * sizeof(double));
    hipMallocManaged(&posi_x_mean, data_p * sizeof(double));
    hipblasCreate(&handle);
    hipblasDscal(handle, data_p, &tmp1, posi_x_mean, 1);
    hipblasDscal(handle, data_p, &tmp2, nega_x_mean, 1);
    hipDeviceSynchronize();
    double prob_p = posi_t / (data_n * 1.0), eta_t, t_eval;
    memset(re_wt, 0, sizeof(double) * data_p);
    memset(re_wt_bar, 0, sizeof(double) * data_p);
    *re_len_auc = 0;
    for (int t = 1; t <= (para_num_passes * data_n); t++) {
        const int *xt_inds = x_tr_inds + x_tr_poss[(t - 1) % data_n]; // receive zt=(xt,yt)
        const double *xt_vals = x_tr_vals + x_tr_poss[(t - 1) % data_n];
        eta_t = para_c / sqrt(t); // current learning rate
        //a_wt = cblas_ddot(data_p, re_wt, 1, posi_x_mean, 1); // update a(wt)
        //b_wt = cblas_ddot(data_p, re_wt, 1, nega_x_mean, 1); // para_b(wt)
        hipMallocManaged(&re_wt, data_p * sizeof(double));
        hipblasDdot(handle, data_p, re_wt, 1, posi_x_mean, 1, &a_wt);
        hipblasDdot(handle, data_p, re_wt, 1, nega_x_mean, 1, &b_wt);
        alpha_wt = b_wt - a_wt; // alpha(wt)
        double wt_dot = 0.0;
        for (int tt = 0; tt < x_tr_lens[(t - 1) % data_n]; tt++)
            wt_dot += (re_wt[xt_inds[tt]] * xt_vals[tt]);
        double weight = data_y_tr[(t - 1) % data_n] > 0 ?
                        2. * (1.0 - prob_p) * (wt_dot - a_wt) -
                        2. * (1.0 + alpha_wt) * (1.0 - prob_p) :
                        2.0 * prob_p * (wt_dot - b_wt) + 2.0 * (1.0 + alpha_wt) * prob_p;
        for (int tt = 0; tt < x_tr_lens[(t - 1) % data_n]; tt++) // gradient descent
            re_wt[xt_inds[tt]] += -eta_t * weight * xt_vals[tt];
        if (para_reg_opt == 0) { // elastic-net
            double tmp_demon = (eta_t * para_l2_reg + 1.);
            for (int k = 0; k < data_p; k++) {
                double tmp_sign = (double) sign(re_wt[k]) / tmp_demon;
                re_wt[k] = tmp_sign * fmax(0.0, fabs(re_wt[k]) - eta_t * para_l1_reg);
            }
        } else { // l2-regularization
            //cblas_dscal(data_p, 1. / (eta_t * para_l2_reg + 1.), re_wt, 1);
            double tmp3 = 1. / (eta_t * para_l2_reg + 1.);
            hipblasDscal(handle, data_p, &tmp3, re_wt, 1);
        }
        double tmp4 = 1.0;
        //cblas_daxpy(data_p, 1., re_wt, 1, re_wt_bar, 1);
        hipblasDaxpy(handle, data_p, &tmp4, re_wt, 1, re_wt_bar, 1);
        if ((fmod(t, para_step_len) == 1.)) { // evaluate the AUC score
            t_eval = clock();
            for (int q = 0; q < data_n; q++) {
                xt_inds = x_tr_inds + x_tr_poss[q];
                xt_vals = x_tr_vals + x_tr_poss[q];
                y_pred[q] = 0.0;
                for (int tt = 0; tt < x_tr_lens[q]; tt++)
                    y_pred[q] += re_wt[xt_inds[tt]] * xt_vals[tt];
            }
            re_auc[*re_len_auc] = _auc_score(data_y_tr, y_pred, data_n);
            re_rts[(*re_len_auc)++] = clock() - start_time - (clock() - t_eval);
        }
    }
    double tmp5 = 1. / (para_num_passes * data_n);
    // cblas_dscal(data_p, 1. / (para_num_passes * data_n), re_wt_bar, 1);
    hipblasDscal(handle, data_p, &tmp5, re_wt_bar, 1);
    double tmp6 = 1. / CLOCKS_PER_SEC;
    // cblas_dscal(*(re_len_auc), 1. / CLOCKS_PER_SEC, re_rts, 1);
    hipblasDscal(handle, *(re_len_auc), &tmp6, re_rts, 1);
    free(y_pred);
    free(nega_x_mean);
    free(posi_x_mean);
    free(grad_wt);
    hipblasDestroy(handle);
}
